#include "hip/hip_runtime.h"
#include "gtom/include/Prerequisites.cuh"
#include "gtom/include/Angles.cuh"
#include "gtom/include/Helper.cuh"


namespace gtom
{
	////////////////////////////
	//CUDA kernel declarations//
	////////////////////////////

	//Kernel for applying an irregular spherical mask to the input
	template <class T, int ndims> __global__ void IrregularSphereMaskKernel(T* d_input, T* d_output, int3 dims, tfloat sigma, tfloat3 center
#if __CUDACC_VER_MAJOR__ >= 12
		, hipTextureObject_t texIrregularSphereRadius2d_obj
#endif
	);

	///////////
	//Globals//
	///////////

#if __CUDACC_VER_MAJOR__ < 12
	static texture<tfloat, 2, hipReadModeElementType> texIrregularSphereRadius2d;  //added static in order to allow parallel builds
	#define TEX_RADIUS(phi, theta) tex2D(texIrregularSphereRadius2d, phi, theta)
#else
	#define TEX_RADIUS(phi, theta) tex2D<tfloat>(texIrregularSphereRadius2d_obj, phi, theta)
#endif

	////////////////
	//Host methods//
	////////////////

	template <class T> void d_IrregularSphereMask(T* d_input,
		T* d_output,
		int3 dims,
		tfloat* d_radiusmap,
		int2 anglesteps,
		tfloat sigma,
		tfloat3* center,
		int batch)
	{
		tfloat* d_pitched = NULL;
		int pitchedwidth = anglesteps.x * sizeof(tfloat);
		d_pitched = (tfloat*)CudaMallocAligned2D(anglesteps.x * sizeof(tfloat), anglesteps.y, &pitchedwidth);
		for (int y = 0; y < anglesteps.y; y++)
			hipMemcpy((char*)d_pitched + y * pitchedwidth,
				d_radiusmap + y * anglesteps.x,
				anglesteps.x * sizeof(tfloat),
				hipMemcpyDeviceToDevice);

#if __CUDACC_VER_MAJOR__ < 12
		texIrregularSphereRadius2d.normalized = true;
		texIrregularSphereRadius2d.filterMode = hipFilterModeLinear;
		texIrregularSphereRadius2d.addressMode[0] = hipAddressModeMirror;
		texIrregularSphereRadius2d.addressMode[1] = hipAddressModeMirror;

		hipChannelFormatDesc desc = hipCreateChannelDesc<tfloat>();
		hipBindTexture2D(NULL,
			texIrregularSphereRadius2d,
			d_pitched,
			desc,
			anglesteps.x,
			anglesteps.y,
			pitchedwidth);
#else
		//CUDA >=12 texture object setup
		hipResourceDesc resDesc{};
		resDesc.resType = hipResourceTypePitch2D;
		resDesc.res.pitch2D.devPtr = d_pitched;
		resDesc.res.pitch2D.pitchInBytes = pitchedwidth;
		resDesc.res.pitch2D.width = anglesteps.x;
		resDesc.res.pitch2D.height = anglesteps.y;
		resDesc.res.pitch2D.desc = hipCreateChannelDesc<tfloat>();

		hipTextureDesc texDesc{};
		texDesc.addressMode[0] = hipAddressModeMirror;
		texDesc.addressMode[1] = hipAddressModeMirror;
		texDesc.filterMode = hipFilterModeLinear;
		texDesc.readMode = hipReadModeElementType;
		texDesc.normalizedCoords = 1;

		hipTextureObject_t texIrregularSphereRadius2d_obj;
		hipCreateTextureObject(&texIrregularSphereRadius2d_obj, &resDesc, &texDesc, nullptr);
#endif

		tfloat3 _center = center != NULL ? *center : tfloat3(dims.x / 2, dims.y / 2, dims.z / 2);

		int TpB = min(NextMultipleOf(dims.x, 32), 256);
		dim3 grid = dim3(dims.y, dims.z, batch);
		if (DimensionCount(dims) <= 2)
			IrregularSphereMaskKernel<T, 2> << <grid, TpB >> > (d_input, d_output, dims, sigma, _center
#if __CUDACC_VER_MAJOR__ >= 12
				, texIrregularSphereRadius2d_obj
#endif
				);
		else
			IrregularSphereMaskKernel<T, 3> << <grid, TpB >> > (d_input, d_output, dims, sigma, _center
#if __CUDACC_VER_MAJOR__ >= 12
				, texIrregularSphereRadius2d_obj
#endif
				);

#if __CUDACC_VER_MAJOR__ < 12
		//Unbind texture
		hipUnbindTexture(texIrregularSphereRadius2d);
#else
		//Destroy texture object
		hipDestroyTextureObject(texIrregularSphereRadius2d_obj);
#endif
		hipFree(d_pitched);
	}

	template void d_IrregularSphereMask<tfloat>(tfloat* d_input, tfloat* d_output, int3 dims, tfloat* d_radiusmap, int2 anglesteps, tfloat sigma, tfloat3* center, int batch);

	////////////////
	//CUDA kernels//
	////////////////

	template <class T, int ndims> __global__ void IrregularSphereMaskKernel(T* d_input, T* d_output, int3 dims, tfloat sigma, tfloat3 center
#if __CUDACC_VER_MAJOR__ >= 12
		, hipTextureObject_t texIrregularSphereRadius2d_obj
#endif
	)
	{
		if (threadIdx.x >= dims.x)
			return;

		//For batch mode
		int offset = blockIdx.z * Elements(dims) + blockIdx.y * dims.x * dims.y + blockIdx.x * dims.x;

		int x, y, z;
		float length;
		T maskvalue;

		//Squared y and z distance from center
		y = blockIdx.x - center.y;
		if (ndims > 2)
			z = blockIdx.y - center.z;
		else
			z = 0;

		for (int idx = threadIdx.x; idx < dims.x; idx += blockDim.x)
		{
			x = idx - center.x;

			length = sqrt((float)(x * x + y * y + z * z));

			glm::vec3 direction((float)x / length, (float)y / length, (float)z / length);
			float theta = acos((float)(-direction.x));
			float phi = atan2((float)direction.y / sin(theta), (float)direction.z / sin(theta));

			tfloat radius = TEX_RADIUS(phi, theta);

			if (length < radius)
				maskvalue = 1;
			else
			{
				//Smooth border
				if (sigma > (tfloat)0)
				{
					maskvalue = exp(-((length - radius) * (length - radius) / (sigma * sigma)));
					if (maskvalue < (tfloat)0.1353)
						maskvalue = 0;
				}
				//Hard border
				else
					maskvalue = max((T)1 - (length - radius), (T)0);
			}

			//Write masked input to output
			d_output[offset + idx] = maskvalue * d_input[offset + idx];
			//d_output[offset + idx] = radius;		
		}
	}
}
